#include <cstdio>
#include <hip/hip_runtime.h>

int main(int argc, char **argv) {
  int nElem = 1024;
  dim3 block(1024);
  dim3 grid((nElem - 1) / block.x + 1);
  printf("grid.x is %d, block.x is %d\n", grid.x, block.x);

  block.x = 512;
  grid.x = ((nElem - 1) / block.x + 1);
  printf("grid.x is %d, block.x is %d\n", grid.x, block.x);

  hipDeviceReset();
  return 0;
}
