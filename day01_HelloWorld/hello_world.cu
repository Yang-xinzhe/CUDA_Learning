
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdio.h>

__global__ void Hello_World(void) { printf("GPU: Hello World\n"); }

int main() {
  printf("CPU: Hello World\n");

  Hello_World<<<1, 1>>>();
  hipDeviceReset(); // if no this line, it can not out put hello world from GPU
  return 0;
}
