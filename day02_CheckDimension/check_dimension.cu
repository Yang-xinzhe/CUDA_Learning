#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void checkIndex(void) {
  printf("threadIdx:(%d, %d, %d), blockIdx:(%d, %d, %d), blockDim(%d, %d, %d), "
         "gridDim:(%d, %d, %d)\n",
         threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y,
         blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y,
         gridDim.z);
}

int main() {
  int nElem = 6;
  dim3 block(5);
  dim3 grid((nElem + block.x - 1) / block.x);

  printf("grid.x is %d, grid.y is %d, grid.z is %d\n", grid.x, grid.y, grid.z);
  printf("block.x is %d, block.y is %d, block.z is %d\n", block.x, block.y,
         block.z);

  checkIndex<<<grid, block>>>();
  hipDeviceReset();
  return 0;
}
